
//compute global vector sum C = A +B
// Each thread performs one pair-wise addiction

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ 
void vecAddKernel(float* A, float* B, float*C, int n) {
    // Perform sum value in each tread
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n ) {
        C[i] = A[i] + B[i];
    }
}

void vecAdd(float* A, float* B, float* C, int n) {
    // In device
    // Create device vectors 
    float *A_d, *B_d, *C_d;
    int size = n * sizeof(float);

    // Alocate mmemory for device
    hipMalloc((void **) &A_d, size);
    hipMalloc((void **) &B_d, size);
    hipMalloc((void **) &C_d, size);

    // Copy host memory values to device memory
    hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);
    // Sum
    vecAddKernel<<<ceil(n/255.0), 255>>>(A_d, B_d, C_d, n);

    // Copy device values to host memory
    hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main(void) {
    // create host vectors
	float *A_h, *B_h, *C_h;
	int n = 800;
	// allocate memory for host vectors
	A_h = (float*)malloc(sizeof(float)*n);
	B_h = (float*)malloc(sizeof(float)*n);
	C_h = (float*)malloc(sizeof(float)*n);

	for (int i = 0; i < n; i++) {
		A_h[i] = (10);
		B_h[i] = (10);
	}

    vecAdd(A_h, B_h, C_h, n);

    // free host memmory
    free(A_h);
    free(A_h);
    free(C_h);


}